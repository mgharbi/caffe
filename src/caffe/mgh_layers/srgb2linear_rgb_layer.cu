#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>
#include <cmath>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/mgh_layers/srgb2linear_rgb_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SRGB2LinearRGBForward(const int n, const Dtype* in, Dtype* out,
    int height, int width, int chans)
{
    CUDA_KERNEL_LOOP(index, n) {
        Dtype px = in[index];
        if ( px > 0.04045 ) {
            px = pow(( px + 0.055 ) / 1.055 , 2.4);
        } else {
            px = px / 12.92;
        }  
        out[index] = px;
    }
}

template <typename Dtype>
__global__ void SRGB2LinearRGBBackward(const int n, const Dtype* top_diff, Dtype* bottom_diff, const Dtype* bottom_data,
    int height, int width, int chans)
{
    CUDA_KERNEL_LOOP(index, n) {
        Dtype dx = top_diff[index];

        Dtype px = bottom_data[index];

        Dtype dr = Dtype(0);

        if ( px > 0.04045) {
            dr = 2.4/1.055*pow(( px + 0.055 ) / 1.055  , 2.4-1);
        } else {
            dr = 1.0/12.92;
        }  

        bottom_diff[index] = dr*dx;
    }
}

template <typename Dtype>
void SRGB2LinearRGBLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();

    vector<int> shape = bottom[0]->shape();
    int npix = shape[0]*shape[1]*shape[2]*shape[3];
    SRGB2LinearRGBForward<Dtype><<<CAFFE_GET_BLOCKS(npix), CAFFE_CUDA_NUM_THREADS>>>(
            npix, bottom_data, top_data, 
            shape[3], shape[2],shape[1]);
    CUDA_POST_KERNEL_CHECK;

}

template <typename Dtype>
void SRGB2LinearRGBLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
  const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

    const Dtype* top_diff    = top[0]->cpu_diff();
    const Dtype* bottom_data = bottom[0]->cpu_data();
    Dtype* bottom_diff       = bottom[0]->mutable_cpu_diff();

    vector<int> shape = bottom[0]->shape();
    int npix = shape[0]*shape[1]*shape[2]*shape[3];
    SRGB2LinearRGBBackward<Dtype><<<CAFFE_GET_BLOCKS(npix), CAFFE_CUDA_NUM_THREADS>>>(
            npix, top_diff, bottom_diff, bottom_data, 
            shape[3], shape[2],shape[1]);
    CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(SRGB2LinearRGBLayer);

}  // namespace caffe
