#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>
#include <cmath>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/mgh_layers/srgb2linear_rgb_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SRGB2LinearRGBForward(const int n, const Dtype* in, Dtype* out,
    int height, int width, int chans)
{
    float t_ = 0.04045 ;
    float a_ = 0.055;
    float b_ = 1+a_;
    float c_ = 2.4;
    float d_ = 12.92;
    CUDA_KERNEL_LOOP(index, n) {
        Dtype px = in[index];
        if ( px > t_ ) {
            px = pow(( px + a_ ) / b_ , c_);
        } else {
            px = px / d_;
        }  
        out[index] = px;
    }
}

template <typename Dtype>
__global__ void SRGB2LinearRGBBackward(const int n, const Dtype* top_diff, Dtype* bottom_diff, const Dtype* bottom_data,
    int height, int width, int chans)
{
    float t_ = 0.04045 ;
    float a_ = 0.055;
    float b_ = 1+a_;
    float c_ = 2.4;
    float d_ = 12.92;
    CUDA_KERNEL_LOOP(index, n) {
        Dtype dx = top_diff[index];

        Dtype px = bottom_data[index];

        Dtype dr = Dtype(0);

        if ( px > t_) {
            dr = c_/b_*pow(( px + a_ ) / b_  , c_-1);
        } else {
            dr = 1.0/d_;
        }  

        bottom_diff[index] = dr*dx;
    }
}

template <typename Dtype>
void SRGB2LinearRGBLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();

    vector<int> shape = bottom[0]->shape();
    int npix = shape[0]*shape[1]*shape[2]*shape[3];
    SRGB2LinearRGBForward<Dtype><<<CAFFE_GET_BLOCKS(npix), CAFFE_CUDA_NUM_THREADS>>>(
            npix, bottom_data, top_data, 
            shape[3], shape[2],shape[1]);
    CUDA_POST_KERNEL_CHECK;

}

template <typename Dtype>
void SRGB2LinearRGBLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
  const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

    const Dtype* top_diff    = top[0]->cpu_diff();
    const Dtype* bottom_data = bottom[0]->cpu_data();
    Dtype* bottom_diff       = bottom[0]->mutable_cpu_diff();

    vector<int> shape = bottom[0]->shape();
    int npix = shape[0]*shape[1]*shape[2]*shape[3];
    SRGB2LinearRGBBackward<Dtype><<<CAFFE_GET_BLOCKS(npix), CAFFE_CUDA_NUM_THREADS>>>(
            npix, top_diff, bottom_diff, bottom_data, 
            shape[3], shape[2],shape[1]);
    CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(SRGB2LinearRGBLayer);

}  // namespace caffe
