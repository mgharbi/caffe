#include "hip/hip_runtime.h"
#include <cfloat>
#include <cmath>

#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/mgh_layers/randomize_hsv_layer.hpp"
#include "caffe/util/rng.hpp"
#include "caffe/util/io.hpp"

namespace caffe {

template <typename Dtype>
__global__ void RandomizeHSVForward(const int n, const Dtype* in, Dtype* out,
    Dtype rand_h, Dtype rand_s, Dtype rand_v,
    int chans, int height, int width)
{
    CUDA_KERNEL_LOOP(index, n) {
        int blob_idx = index / (height*width);
        int px = index % (height*width);
        int src_idx = blob_idx * height*width*chans // blob n index
            + px; // pixel index
        int chan_stride = height*width;

        Dtype h = in[src_idx];
        Dtype s = in[src_idx + chan_stride];
        Dtype v = in[src_idx + 2*chan_stride];

        out[src_idx+0*chan_stride] = fmod(Dtype(h+rand_h+1.0),Dtype(1.0));
        out[src_idx+1*chan_stride] = fmin(fmax(Dtype(s+rand_s),Dtype(0.0)), Dtype(1.0));
        out[src_idx+2*chan_stride] = fmin(fmax(Dtype(v+rand_v),Dtype(0.0)), Dtype(1.0));
    }
}

template <typename Dtype>
void RandomizeHSVLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();

    const int count = top[0]->count();

    // Process only during training
    float randomize;
    caffe_rng_uniform<float>(1, 0.0, 1.0, &randomize);
    if (randomize < randomized_ratio_ || this->phase_ != TRAIN) {
        caffe_copy(count, bottom_data, top_data);
        return;
    }

    vector<int> shape = bottom[0]->shape();

    Dtype rand_h;
    Dtype rand_s;
    Dtype rand_v;
    caffe_rng_uniform<Dtype>(1, 0.0, 1.0, &rand_h);
    caffe_rng_uniform<Dtype>(1, -0.3, 0.3, &rand_s);
    caffe_rng_uniform<Dtype>(1, -0.3, 0.3, &rand_v);
    int npix = shape[0]*shape[2]*shape[3];
    RandomizeHSVForward<Dtype><<<CAFFE_GET_BLOCKS(npix), CAFFE_CUDA_NUM_THREADS>>>(
            npix, bottom_data, top_data, 
            rand_h, rand_s, rand_v,
            shape[1], shape[2],shape[3]);
    CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void RandomizeHSVLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
  const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    NOT_IMPLEMENTED;
}

INSTANTIATE_LAYER_GPU_FUNCS(RandomizeHSVLayer);

}  // namespace caffe
