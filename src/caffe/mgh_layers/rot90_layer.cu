#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/mgh_layers/rot90_layer.hpp"
#include "caffe/util/rng.hpp"

namespace caffe {

template <typename Dtype>
__global__ void Rot90Forward(const int n, const Dtype* in, Dtype* out, const float * randomize,
    int chans, int height, int width)
{
    CUDA_KERNEL_LOOP(index, n) {
        int blob_idx = index / (height*width*chans);

        int vx = index % (height*width*chans);
        int z  = vx / (height*width);
        int px = vx % (height*width);
        int y  = px/width;
        int x  = px % width;

        int xp = x;
        int yp = y;

        if(randomize[blob_idx] < 0.25f) { // angle 0
            xp = x;
            yp = y;
        } else if(randomize[blob_idx] < 0.5f) { // angle pi/2
            xp = height-1-y;
            yp = x;
        } else if(randomize[blob_idx] < 0.75f) { // angle pi
            xp = width-1-x;
            yp = height-1-y;
        } else { // angle 3*pi/2
            xp = width-1-x;
            yp = y;
        }
        int src_idx = xp + width*(yp + height * (z + chans*blob_idx));

        // Flip only half of the images on average
        int dst_idx = x + width*(y + height * (z + chans*blob_idx));
        out[dst_idx] = in[src_idx];
    }
}

template <typename Dtype>
void Rot90Layer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();

    vector<int> shape = bottom[0]->shape();
    int w = shape[3];

    vector<int> rand_shape(1);
    rand_shape[0] = shape[0];
    Blob<float> randomize(rand_shape);
    caffe_rng_uniform<float>(shape[0], 0.0, 1.0, randomize.mutable_cpu_data());

    int npix = shape[0]*shape[1]*shape[2]*shape[3];
    Rot90Forward<Dtype><<<CAFFE_GET_BLOCKS(npix), CAFFE_CUDA_NUM_THREADS>>>(
            npix, bottom_data, top_data, randomize.gpu_data() ,
            shape[1], shape[2], shape[3]);
    CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(Rot90Layer);

}  // namespace caffe
