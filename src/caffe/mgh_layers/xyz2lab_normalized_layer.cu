#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>
#include <cmath>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/mgh_layers/xyz2lab_normalized_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void XYZ2LABNormalizedForward(const int n, const Dtype* in, Dtype* out,
    double X_ref, double Y_ref, double Z_ref,
    int height, int width, int chans)
{
    CUDA_KERNEL_LOOP(index, n) {
        int blob_idx = index / (height*width);
        int px = index % (height*width);
        int src_idx = blob_idx * height*width*chans // blob n index
            + px; // pixel index
        int chan_stride = height*width;

        Dtype X = in[src_idx + 0*chan_stride];
        Dtype Y = in[src_idx + 1*chan_stride];
        Dtype Z = in[src_idx + 2*chan_stride];

        X  = X/X_ref;
        Y  = Y/Y_ref;
        Z  = Z/Z_ref;

        if ( X > 0.008856 ){
            X = pow(Dtype(X),  Dtype(1.0/3.0) );
        } else {
            X = ( 7.787 * X ) + ( 16.0 / 116.0 );
        }   
        if ( Y > 0.008856 ){
            Y = pow(Dtype(Y),  Dtype(1.0/3.0) );
        } else {
            Y = ( 7.787 * Y ) + ( 16.0 / 116.0 );
        }   
        if ( Z > 0.008856 ){
            Z = pow(Dtype(Z),  Dtype(1.0/3.0) );
        } else {
            Z = ( 7.787 * Z ) + ( 16.0 / 116.0 );
        }   

        double L = 116.0*Y - 16.0;
        double a = 500.0*(X-Y);
        double b = 200.0*(Y-Z);

        out[src_idx+0*chan_stride] = L/100.0;
        out[src_idx+1*chan_stride] = (a+127.0)/255.0;
        out[src_idx+2*chan_stride] = (b+127.0)/255.0;
    }
}

template <typename Dtype>
__global__ void XYZ2LABNormalizedBackward(const int n, const Dtype* top_diff, Dtype* bottom_diff, const Dtype* bottom_data,
    double X_ref, double Y_ref, double Z_ref,
    int height, int width, int chans)
{
    CUDA_KERNEL_LOOP(index, n) {
        int blob_idx = index / (height*width);
        int px = index % (height*width);
        int src_idx = blob_idx * height*width*chans // blob n index
            + px; // pixel index
        int chan_stride = height*width;

        Dtype dL = top_diff[src_idx + 0*chan_stride]/100.0;
        Dtype da = top_diff[src_idx + 1*chan_stride]/255.0;
        Dtype db = top_diff[src_idx + 2*chan_stride]/255.0;

        Dtype dX_pp =            500.0*da           ;
        Dtype dY_pp = 116.0*dL - 500.0*da + 200.0*db;
        Dtype dZ_pp =                     - 200.0*db;

        Dtype X = bottom_data[src_idx + 0*chan_stride];
        Dtype Y = bottom_data[src_idx + 1*chan_stride];
        Dtype Z = bottom_data[src_idx + 2*chan_stride];

        Dtype Xp = X / X_ref;
        Dtype Yp = Y / Y_ref;
        Dtype Zp = Z / Z_ref;

        Dtype dX_p = 0;
        Dtype dY_p = 0;
        Dtype dZ_p = 0;

        if ( Xp > 0.008856 ){
            dX_p = 1.0/3.0 * pow(Dtype(Xp),  Dtype(-2.0/3.0) );
        } else {
            dX_p = 7.787;
        }   
        if ( Yp > 0.008856 ){
            dY_p = 1.0/3.0 * pow(Dtype(Yp),  Dtype(-2.0/3.0) );
        } else {
            dY_p = 7.787;
        }   
        if ( Zp > 0.008856 ){
            dZ_p = 1.0/3.0 * pow(Dtype(Zp),  Dtype(-2.0/3.0) );
        } else {
            dZ_p = 7.787;
        }   

        bottom_diff[src_idx+0*chan_stride] = dX_p*dX_pp/X_ref;
        bottom_diff[src_idx+1*chan_stride] = dY_p*dY_pp/Y_ref;
        bottom_diff[src_idx+2*chan_stride] = dZ_p*dZ_pp/Z_ref;
    }
}

template <typename Dtype>
void XYZ2LABNormalizedLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();

    vector<int> shape = bottom[0]->shape();
    int npix = shape[0]*shape[2]*shape[3];
    XYZ2LABNormalizedForward<Dtype><<<CAFFE_GET_BLOCKS(npix), CAFFE_CUDA_NUM_THREADS>>>(
            npix, bottom_data, top_data, 
            X_ref, Y_ref, Z_ref,
            shape[3], shape[2],shape[1]);
    CUDA_POST_KERNEL_CHECK;

}

template <typename Dtype>
void XYZ2LABNormalizedLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
  const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

    const Dtype* top_diff    = top[0]->cpu_diff();
    const Dtype* bottom_data = bottom[0]->cpu_data();
    Dtype* bottom_diff       = bottom[0]->mutable_cpu_diff();

    vector<int> shape = bottom[0]->shape();
    int npix = shape[0]*shape[2]*shape[3];
    XYZ2LABNormalizedBackward<Dtype><<<CAFFE_GET_BLOCKS(npix), CAFFE_CUDA_NUM_THREADS>>>(
            npix, top_diff, bottom_diff, bottom_data, 
            X_ref, Y_ref, Z_ref,
            shape[3], shape[2],shape[1]);
    CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(XYZ2LABNormalizedLayer);

}  // namespace caffe
