#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/mgh_layers/flipLR_layer.hpp"
#include "caffe/util/rng.hpp"

namespace caffe {

template <typename Dtype>
__global__ void FlipLRForward(const int n, const Dtype* in, Dtype* out, const float * randomize,
    int chans, int height, int width)
{
    CUDA_KERNEL_LOOP(index, n) {
        int blob_idx = index / (height*width*chans);

        int vx = index % (height*width*chans);
        int z = vx / (height*width);
        int px = vx % (height*width);
        int y = px/width;
        int x = px % width;

        int dst_idx = x + width*(y + height * (z + chans*blob_idx));

        // Flip only half of the images on average
        float f = randomize[0];
        int src_idx = width-1-x + width*(y + height * (z + chans*blob_idx));
        out[dst_idx] = f > 0.5f ? in[dst_idx] : in[src_idx];
    }
}

template <typename Dtype>
void FlipLRLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();

    vector<int> shape = bottom[0]->shape();
    int w = shape[3];

    vector<int> rand_shape(1);
    rand_shape[0] = shape[0];
    Blob<float> randomize(rand_shape);
    caffe_rng_uniform<float>(shape[0], 0.0, 1.0, randomize.mutable_cpu_data());

    LOG(INFO) << "randomize " << randomize.cpu_data()[0];

    int npix = shape[0]*shape[1]*shape[2]*shape[3];
    FlipLRForward<Dtype><<<CAFFE_GET_BLOCKS(npix), CAFFE_CUDA_NUM_THREADS>>>(
            npix, bottom_data, top_data, randomize.gpu_data() ,
            shape[1], shape[2], shape[3]);
    CUDA_POST_KERNEL_CHECK;
    hipDeviceSynchronize();
    // delete[] randomize;
}

INSTANTIATE_LAYER_GPU_FUNCS(FlipLRLayer);

}  // namespace caffe
