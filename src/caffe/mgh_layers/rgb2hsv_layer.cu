#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>
#include <cmath>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/mgh_layers/rgb2hsv_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void RGB2HSVForward(const int n, const Dtype* in, Dtype* out,
    int height, int width, int chans)
{
    CUDA_KERNEL_LOOP(index, n) {
        int blob_idx = index / (height*width);
        int px = index % (height*width);
        int src_idx = blob_idx * height*width*chans // blob n index
            + px; // pixel index
        int chan_stride = height*width;

        Dtype r = in[src_idx + 0*chan_stride];
        Dtype g = in[src_idx + 1*chan_stride];
        Dtype b = in[src_idx + 2*chan_stride];

        Dtype maxi = fmax(r,fmax(g,b));
        Dtype mini = fmin(r,fmin(g,b));
        Dtype delta = maxi-mini;

        Dtype h = 0;
        Dtype v = 0; 
        Dtype s = 0; 

        // Value
        v = maxi;

        // Saturation
        if(v == 0 || delta == 0) {
            s = 0;
        }else{
            s = delta/v;
        }

        // Hue
        if(delta == 0){
            h = 0;
        }else {
            if(maxi == r) {
                h = (g-b)/delta;
            }else if(maxi == g) {
                h = 2.0 + (b-r)/delta;
            }else if(maxi == b) {
                h = 4.0 + (r-g)/delta;
            }
            h = fmod((h+6)/6.0,1.0);
        }

        out[src_idx+0*chan_stride] = h;
        out[src_idx+1*chan_stride] = s;
        out[src_idx+2*chan_stride] = v;
    }
}

template <typename Dtype>
void RGB2HSVLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();

    vector<int> shape = bottom[0]->shape();
    int npix = shape[0]*shape[2]*shape[3];
    RGB2HSVForward<Dtype><<<CAFFE_GET_BLOCKS(npix), CAFFE_CUDA_NUM_THREADS>>>(
            npix, bottom_data, top_data, 
            shape[3], shape[2],shape[1]);
    CUDA_POST_KERNEL_CHECK;

}

template <typename Dtype>
void RGB2HSVLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
  const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    NOT_IMPLEMENTED;
}

INSTANTIATE_LAYER_GPU_FUNCS(RGB2HSVLayer);

}  // namespace caffe
