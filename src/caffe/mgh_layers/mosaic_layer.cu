#include "hip/hip_runtime.h"
#include <cfloat>

#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/mgh_layers/mosaic_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void mosaick_row_kernel(const int n, const int height, const int width,
    const int src_outer_stride, const int src_inner_stride,
    const int dst_outer_stride, const int dst_inner_stride,
    bool store_pattern,
    const Dtype* src, Dtype* dst) 
{

    CUDA_KERNEL_LOOP(index, n) {
        int blob_idx = index / (height);
        int y = index % (height);
        int src_start = blob_idx * src_outer_stride // blob n index
            + y * src_inner_stride; // line index
        int dst_start = blob_idx * dst_outer_stride // blob n index
            + y * dst_inner_stride; // line index

        int chan_stride = height*width;

        if(y % 2 == 0) {
            for (int x = 0; x < width; ++x) {
                if ( x % 2 == 0) { // G
                    // top_data[top[0]->offset(n,0,y,x)] = bottom_data[bottom[0]->offset(n,1,y,x)];
                    dst[dst_start + x] = src[src_start + x + 1*chan_stride];
                    if(store_pattern){
                        // top_data[top[0]->offset(n,2,y,x)] = 1;
                        dst[dst_start + x + 2*chan_stride] = 1;
                    }
                } else { // R
                    // top_data[top[0]->offset(n,0,y,x)] = bottom_data[bottom[0]->offset(n,0,y,x)];
                    dst[dst_start + x] = src[src_start + x + 0*chan_stride];
                    if(store_pattern){
                        // top_data[top[0]->offset(n,1,y,x)] = 1;
                        dst[dst_start + x + 1*chan_stride] = 1;
                    }
                }
            }
        } else {
            for (int x = 0; x < width; ++x) {
                if ( x % 2 == 0) { // B
                    // top_data[top[0]->offset(n,0,y,x)] = bottom_data[bottom[0]->offset(n,2,y,x)];
                    dst[dst_start + x] = src[src_start + x + 2*chan_stride];
                    if(store_pattern){
                        // top_data[top[0]->offset(n,3,y,x)] = 1;
                        dst[dst_start + x + 3*chan_stride] = 1;
                    }
                } else { // G
                    // top_data[top[0]->offset(n,0,y,x)] = bottom_data[bottom[0]->offset(n,1,y,x)];
                    dst[dst_start + x] = src[src_start + x + 1*chan_stride];
                    if(store_pattern){
                        // top_data[top[0]->offset(n,2,y,x)] = 1;
                        dst[dst_start + x + 2*chan_stride] = 1;
                    }
                }
            }
        }
    }
}

template <typename Dtype>
void MosaicLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) 
{

    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();

    vector<int> shape = bottom[0]->shape();
    const int lines = shape[0]*shape[2]; // n blobs, h lines per blob

    const int src_outer_stride = shape[3]*shape[2]*shape[1]; // skip h*w*c between blobs
    const int src_inner_stride = shape[3]; // skip w pixels between lines

    vector<int> tshape = top[0]->shape();
    const int dst_outer_stride = tshape[3]*tshape[2]*tshape[1];
    const int dst_inner_stride = tshape[3];

    mosaick_row_kernel<<<CAFFE_GET_BLOCKS(lines), CAFFE_CUDA_NUM_THREADS>>>(
            lines, top[0]->height(), top[0]->width(),
            src_outer_stride, src_inner_stride,
            dst_outer_stride, dst_inner_stride,
            store_pattern_,
            bottom_data, top_data);
    CUDA_POST_KERNEL_CHECK;
    // Mosaick
    // G R G R G
    // B G B G B
    // G R G R G
}


INSTANTIATE_LAYER_GPU_FUNCS(MosaicLayer);

}  // namespace caffe

