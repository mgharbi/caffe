#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>
#include <cmath>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/mgh_layers/linear_rgb2srgb_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void LinearRGB2SRGBForward(const int n, const Dtype* in, Dtype* out,
    int height, int width, int chans)
{
    const float t_ = 0.0031308;
    const float a_ = 0.055;
    const float b_ = 1+a_;
    const float c_ = 1.0/2.4;
    const float d_ = 12.92;
    CUDA_KERNEL_LOOP(index, n) {
        Dtype px = in[index];
        if ( px > t_ ) {
            px = b_*pow(px,c_) - a_;
        } else {
            px = px * d_;
        }  
        out[index] = px;
    }
}

template <typename Dtype>
__global__ void LinearRGB2SRGBBackward(const int n, const Dtype* top_diff, Dtype* bottom_diff, const Dtype* bottom_data,
    int height, int width, int chans)
{
    const float t_ = 0.0031308;
    const float a_ = 0.055;
    const float b_ = 1+a_;
    const float c_ = 1.0/2.4;
    const float d_ = 12.92;
    CUDA_KERNEL_LOOP(index, n) {
        Dtype dx = top_diff[index];

        Dtype px = bottom_data[index];

        Dtype dr = Dtype(0);

        if ( px > t_ ) {
            dr = b_*c_/pow(px,1-c_) ;
        } else {
            dr = d_;
        }  


        bottom_diff[index] = dr*dx;
    }
}

template <typename Dtype>
void LinearRGB2SRGBLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();

    vector<int> shape = bottom[0]->shape();
    int npix = shape[0]*shape[1]*shape[2]*shape[3];
    LinearRGB2SRGBForward<Dtype><<<CAFFE_GET_BLOCKS(npix), CAFFE_CUDA_NUM_THREADS>>>(
            npix, bottom_data, top_data, 
            shape[3], shape[2],shape[1]);
    CUDA_POST_KERNEL_CHECK;

}

template <typename Dtype>
void LinearRGB2SRGBLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
  const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

    const Dtype* top_diff    = top[0]->cpu_diff();
    const Dtype* bottom_data = bottom[0]->cpu_data();
    Dtype* bottom_diff       = bottom[0]->mutable_cpu_diff();

    vector<int> shape = bottom[0]->shape();
    int npix = shape[0]*shape[1]*shape[2]*shape[3];
    LinearRGB2SRGBBackward<Dtype><<<CAFFE_GET_BLOCKS(npix), CAFFE_CUDA_NUM_THREADS>>>(
            npix, top_diff, bottom_diff, bottom_data, 
            shape[3], shape[2],shape[1]);
    CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(LinearRGB2SRGBLayer);

}  // namespace caffe
