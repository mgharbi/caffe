#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>
#include <cmath>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/mgh_layers/linear_rgb2srgb_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void LinearRGB2SRGBForward(const int n, const Dtype* in, Dtype* out,
    int height, int width, int chans)
{
    CUDA_KERNEL_LOOP(index, n) {
        Dtype px = in[index];

        if ( px > 0.0031308 ) {
            px = 1.055*pow(px,Dtype(1.0/2.4)) - 0.055;
        } else {
            px = px * 12.92;
        }  

        out[index] = px;
    }
}

template <typename Dtype>
__global__ void LinearRGB2SRGBBackward(const int n, const Dtype* top_diff, Dtype* bottom_diff, const Dtype* bottom_data,
    int height, int width, int chans)
{
    CUDA_KERNEL_LOOP(index, n) {
        Dtype dx = top_diff[index];

        Dtype px = bottom_data[index];

        Dtype dr = Dtype(0);

        if ( px > 0.0031308 ) {
            dr = 1.055*(1.0/2.4)*pow(px,Dtype(1.0/2.4-1)) ;
        } else {
            dr = 12.92;
        }  

        bottom_diff[index] = dr*dx;
    }
}

template <typename Dtype>
void LinearRGB2SRGBLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();

    vector<int> shape = bottom[0]->shape();
    int npix = shape[0]*shape[1]*shape[2]*shape[3];
    LinearRGB2SRGBForward<Dtype><<<CAFFE_GET_BLOCKS(npix), CAFFE_CUDA_NUM_THREADS>>>(
            npix, bottom_data, top_data, 
            shape[3], shape[2],shape[1]);
    CUDA_POST_KERNEL_CHECK;

}

template <typename Dtype>
void LinearRGB2SRGBLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
  const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

    const Dtype* top_diff    = top[0]->cpu_diff();
    const Dtype* bottom_data = bottom[0]->cpu_data();
    Dtype* bottom_diff       = bottom[0]->mutable_cpu_diff();

    vector<int> shape = bottom[0]->shape();
    int npix = shape[0]*shape[1]*shape[2]*shape[3];
    LinearRGB2SRGBBackward<Dtype><<<CAFFE_GET_BLOCKS(npix), CAFFE_CUDA_NUM_THREADS>>>(
            npix, top_diff, bottom_diff, bottom_data, 
            shape[3], shape[2],shape[1]);
    CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(LinearRGB2SRGBLayer);

}  // namespace caffe
