#include "hip/hip_runtime.h"
#include <cfloat>

#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/mgh_layers/mosaic_offset_layer.hpp"
#include "caffe/util/rng.hpp"
#include "caffe/util/io.hpp"

#include <math_functions.hpp>

namespace caffe {

template <typename Dtype>
__global__ void MosaicOffsetForward(const int n, const Dtype* in, Dtype* out,
    int height, int width, int chans, const int* offsets)
{
    int offset_x = offsets[0];
    int offset_y = offsets[1];
    CUDA_KERNEL_LOOP(index, n) {
        int blob_idx = index / (height*width*chans);

        int vx = index % (height*width*chans);
        int z  = vx / (height*width);
        int px = vx % (height*width);
        int y  = px/width;
        int x  = px % width;

        int dst_idx = x + width*(y + height * (z + chans*blob_idx));
        int src_idx = max(x-offset_x,0) + width*(max(y-offset_y,0) + height * (z + chans*blob_idx));

        out[dst_idx] = in[src_idx];
    }
}

template <typename Dtype>
void MosaicOffsetLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

    const Dtype* bottom_data = bottom[0]->cpu_data();
    Dtype* top_data = top[0]->mutable_cpu_data();
    const int count = top[0]->count();

    // Process only during training
    if (this->phase_ != TRAIN) {
        caffe_copy(count, bottom_data, top_data);
        return;
    }

    int offset_x = Rand(2);
    int offset_y = Rand(2);

    vector<int> ofshape(1); ofshape[0] = 2;
    Blob<int> offsets(ofshape);
    offsets.mutable_cpu_data()[0] = offset_x;
    offsets.mutable_cpu_data()[1] = offset_y;

    // No offset, copy data over
    if(offset_x == 0 && offset_y == 0) {
        caffe_copy(count, bottom_data, top_data);
        return;
    }

    vector<int> shape = bottom[0]->shape();
    int npix = shape[0]*shape[1]*shape[2]*shape[3];
    MosaicOffsetForward<Dtype><<<CAFFE_GET_BLOCKS(npix), CAFFE_CUDA_NUM_THREADS>>>(
            npix, bottom_data, top_data, 
            shape[3], shape[2],shape[1], offsets.gpu_data());

}
template <typename Dtype>
void MosaicOffsetLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
  const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    NOT_IMPLEMENTED;
}
INSTANTIATE_LAYER_GPU_FUNCS(MosaicOffsetLayer);

}  // namespace caffe
