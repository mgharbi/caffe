#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void DeconvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
        const vector<Blob<Dtype>*>& top) {
    const Dtype* weight = this->blobs_[0]->gpu_data();
    bool doNormalize = true;
    for (int i = 0; i < bottom.size(); ++i) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
        Dtype* top_data = top[i]->mutable_gpu_data();
        for (int n = 0; n < this->num_; ++n) {
            this->backward_gpu_gemm(bottom_data + bottom[i]->offset(n), weight,
                    top_data + top[i]->offset(n));
            if (this->bias_term_) {
                const Dtype* bias = this->blobs_[1]->gpu_data();
                this->forward_gpu_bias(top_data + top[i]->offset(n), bias);
            }
            if(doNormalize){ // Normalize boundaries
                this->normalize_boundaries_gpu(top_data + top[i]->offset(n), top_data + top[i]->offset(n));
            }
        }
    }
}

template <typename Dtype>
void DeconvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
        const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    const Dtype* weight = this->blobs_[0]->gpu_data();
    Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
    for (int i = 0; i < top.size(); ++i) {
        const Dtype* top_diff = top[i]->gpu_diff();
        const Dtype* bottom_data = bottom[i]->gpu_data();
        Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();

        bool doNormalize = true;

        // Bias gradient, if necessary.
        if (this->bias_term_ && this->param_propagate_down_[1]) {
            Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
            vector<int> shape(4);
            shape[0] = 1;
            shape[1] = top[i]->shape(1);
            shape[2] = top[i]->shape(2);
            shape[3] = top[i]->shape(3);
            Blob<Dtype> normalized(shape);
            Dtype * norm_diff = normalized.mutable_gpu_data();
            for (int n = 0; n < this->num_; ++n) {
                if(doNormalize) {
                    this->normalize_boundaries_gpu(top_diff + top[i]->offset(n), norm_diff);
                    this->backward_gpu_bias(bias_diff, norm_diff);
                } else {
                    this->backward_gpu_bias(bias_diff, top_diff + top[i]->offset(n));
                }
            }
        }
        if (this->param_propagate_down_[0] || propagate_down[i]) {
            vector<int> shape(4);
            shape[0] = 1;
            shape[1] = top[i]->shape(1);
            shape[2] = top[i]->shape(2);
            shape[3] = top[i]->shape(3);
            Blob<Dtype> normalized(shape);
            Dtype * norm_diff = normalized.mutable_gpu_data();
            for (int n = 0; n < this->num_; ++n) {
                if(doNormalize) {
                    this->normalize_boundaries_gpu(top_diff + top[i]->offset(n), norm_diff);
                    // gradient w.r.t. weight. Note that we will accumulate diffs.
                    if (this->param_propagate_down_[0]) {
                        this->weight_gpu_gemm(norm_diff,
                                bottom_data + bottom[i]->offset(n), weight_diff);
                    }
                    // gradient w.r.t. bottom data, if necessary.
                    if (propagate_down[i]) {
                        this->forward_gpu_gemm(norm_diff, weight,
                                bottom_diff + bottom[i]->offset(n));
                    }
                } else {
                    // gradient w.r.t. weight. Note that we will accumulate diffs.
                    if (this->param_propagate_down_[0]) {
                        this->weight_gpu_gemm(top_diff + top[i]->offset(n),
                                bottom_data + bottom[i]->offset(n), weight_diff);
                    }
                    // gradient w.r.t. bottom data, if necessary.
                    if (propagate_down[i]) {
                        this->forward_gpu_gemm(top_diff + top[i]->offset(n), weight,
                                bottom_diff + bottom[i]->offset(n));
                    }
                }
            }
        }
    }
}

template <typename Dtype>
__global__ void normalize_boundaries_gpu_kernel(const Dtype* input, Dtype* output,const int n,
    const int height, const int width,
    const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int height_col, const int width_col
)
{
  CUDA_KERNEL_LOOP(index, n) {
    // Dtype val = 0;
    int vcount = 0;
    int w = index % width + pad_w;
    int h = (index / width) % height + pad_h;
    // int c = index / (width * height);
    // compute the start and end of the output
    int w_col_start = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    int w_col_end = min(w / stride_w + 1, width_col);
    int h_col_start = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    int h_col_end = min(h / stride_h + 1, height_col);

    // int offset = (c * kernel_h * kernel_w + h * kernel_w + w) * height_col * width_col;
    // int coeff_h_col = (1 - stride_h * kernel_w * height_col) * width_col;
    // int coeff_w_col = (1 - stride_w * height_col * width_col);
    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        // val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
        vcount += 1;
      }
    }
    // data_im[index] = val;
    if(vcount > 0) {
        output[index] = input[index]/ vcount;
    } else {
        output[index] = input[index];
    }
  }
}

template <typename Dtype>
void DeconvolutionLayer<Dtype>::normalize_boundaries_gpu(const Dtype* input, Dtype* output)
{
    int channels = this->conv_in_channels_;
    int height   = this->conv_in_height_;
    int width    = this->conv_in_width_;
    int pad_h    = this->pad_h_;
    int pad_w    = this->pad_w_;
    int stride_h = this->stride_h_;
    int stride_w = this->stride_w_;
    int kernel_h = this->kernel_h_;
    int kernel_w = this->kernel_w_;

    int height_col = (height + 2 * pad_h - kernel_h) / stride_h + 1;
    int width_col = (width + 2 * pad_w - kernel_w) / stride_w + 1;
    int num_kernels = channels * height * width;
    normalize_boundaries_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
        CAFFE_CUDA_NUM_THREADS>>>(input, output,
                num_kernels, height, width, kernel_h, kernel_w, pad_h,
                pad_w, stride_h, stride_w, height_col, width_col);
    CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(DeconvolutionLayer);

}  // namespace caffe
