#include "hip/hip_runtime.h"
#include <cfloat>
#include <cmath>

#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/mgh_layers.hpp"
#include "caffe/util/rng.hpp"
#include "caffe/util/io.hpp"

namespace caffe {

template <typename Dtype>
__global__ void RandomizeHSVForward(const int n, const Dtype* in, Dtype* out,
    Dtype rand_h, Dtype rand_s, Dtype rand_v,
    int height, int width, int chans)
{
    CUDA_KERNEL_LOOP(index, n) {
        int blob_idx = index / (height*width);
        int px = index % (height*width);
        int src_idx = blob_idx * height*width*chans // blob n index
            + px; // pixel index
        int chan_stride = height*width;

        Dtype h = in[src_idx];
        Dtype s = in[src_idx + chan_stride];
        Dtype v = in[src_idx + 2*chan_stride];

        out[src_idx+0*chan_stride] = fmod(Dtype(h+rand_h+1.0),Dtype(1.0));
        out[src_idx+1*chan_stride] = fmin(fmax(Dtype(s+rand_s),Dtype(0.0)), Dtype(1.0));
        out[src_idx+2*chan_stride] = fmin(fmax(Dtype(v+rand_v),Dtype(0.0)), Dtype(1.0));
    }
}

template <typename Dtype>
void RandomizeHSVLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();

    const int count = top[0]->count();

    // Process only during training
    float randomize;
    caffe_rng_uniform<float>(1, 0.0, 1.0, &randomize);
    LOG(INFO) << "GPU random " << randomized_ratio_;
    if (randomize < randomized_ratio_ || this->phase_ != TRAIN) {
        caffe_copy(count, bottom_data, top_data);
        return;
    }

    vector<int> shape = bottom[0]->shape();

    Dtype rand_h;
    Dtype rand_s;
    Dtype rand_v;
    caffe_rng_uniform<Dtype>(1, 0.0, 1.0, &rand_h);
    caffe_rng_uniform<Dtype>(1, -0.3, 0.3, &rand_s);
    caffe_rng_uniform<Dtype>(1, -0.3, 0.3, &rand_v);
    int npix = shape[0]*shape[2]*shape[3];
    RandomizeHSVForward<Dtype><<<CAFFE_GET_BLOCKS(npix), CAFFE_CUDA_NUM_THREADS>>>(
            npix, bottom_data, top_data, 
            rand_h, rand_s, rand_v,
            shape[3], shape[2],shape[1]);
    CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(RandomizeHSVLayer);

}  // namespace caffe
