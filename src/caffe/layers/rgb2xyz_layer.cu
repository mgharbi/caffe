#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>
#include <cmath>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/mgh_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void RGB2XYZForward(const int n, const Dtype* in, Dtype* out,
    int height, int width, int chans)
{
    CUDA_KERNEL_LOOP(index, n) {
        int blob_idx = index / (height*width);
        int px = index % (height*width);
        int src_idx = blob_idx * height*width*chans // blob n index
            + px; // pixel index
        int chan_stride = height*width;

        Dtype r = in[src_idx + 0*chan_stride];
        Dtype g = in[src_idx + 1*chan_stride];
        Dtype b = in[src_idx + 2*chan_stride];

        if ( r > 0.04045 ) {
            r = pow(Dtype(( r + 0.055 ) / 1.055) , Dtype(2.4));
        } else {
            r = r / 12.92;
        }  
        if ( g > 0.04045 ) {
            g = pow(Dtype(( g + 0.055 ) / 1.055) , Dtype(2.4));
        } else {
            g = g / 12.92;
        }  
        if ( b > 0.04045 ) {
            b = pow(Dtype(( b + 0.055 ) / 1.055) , Dtype(2.4));
        } else {
            b = b / 12.92;
        }  

        //Observer. = 2°, Illuminant = D65
        Dtype X = r * 0.412453 + g * 0.357580 + b * 0.180423;
        Dtype Y = r * 0.212671 + g * 0.715160 + b * 0.072169;
        Dtype Z = r * 0.019334 + g * 0.119193 + b * 0.950227;

        out[src_idx+0*chan_stride] = X;
        out[src_idx+1*chan_stride] = Y;
        out[src_idx+2*chan_stride] = Z;
    }
}

template <typename Dtype>
__global__ void RGB2XYZBackward(const int n, const Dtype* top_diff, Dtype* bottom_diff, const Dtype* bottom_data,
    int height, int width, int chans)
{
    CUDA_KERNEL_LOOP(index, n) {
        int blob_idx = index / (height*width);
        int px = index % (height*width);
        int src_idx = blob_idx * height*width*chans // blob n index
            + px; // pixel index
        int chan_stride = height*width;

        Dtype dx = top_diff[src_idx + 0*chan_stride];
        Dtype dy = top_diff[src_idx + 1*chan_stride];
        Dtype dz = top_diff[src_idx + 2*chan_stride];

        Dtype drp = 0.412453*dx + 0.212671*dy +  0.019334*dz;
        Dtype dgp = 0.357580*dx + 0.715160*dy +  0.119193*dz;
        Dtype dbp = 0.180423*dx + 0.072169*dy +  0.950227*dz;

        Dtype r = bottom_data[src_idx + 0*chan_stride];
        Dtype g = bottom_data[src_idx + 1*chan_stride];
        Dtype b = bottom_data[src_idx + 2*chan_stride];

        Dtype dr = Dtype(0);
        Dtype dg = Dtype(0);
        Dtype db = Dtype(0);

        if ( r > 0.04045 ) {
            dr = 2.4/1.055*pow(( r + 0.055 ) / 1.055 , 1.4);
        } else {
            dr = 1.0 / 12.92;
        }  
        if ( g > 0.04045 ) {
            dg = 2.4/1.055*pow(( g + 0.055 ) / 1.055 , 1.4);
        } else {
            dg = 1.0 / 12.92;
        }  
        if ( b > 0.04045 ) {
            db = 2.4/1.055*pow(( b + 0.055 ) / 1.055 , 1.4);
        } else {
            db = 1.0 / 12.92;
        }  

        bottom_diff[src_idx+0*chan_stride] = dr*drp;
        bottom_diff[src_idx+1*chan_stride] = dg*dgp;
        bottom_diff[src_idx+2*chan_stride] = db*dbp;
    }
}

template <typename Dtype>
void RGB2XYZLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();

    vector<int> shape = bottom[0]->shape();
    int npix = shape[0]*shape[2]*shape[3];
    RGB2XYZForward<Dtype><<<CAFFE_GET_BLOCKS(npix), CAFFE_CUDA_NUM_THREADS>>>(
            npix, bottom_data, top_data, 
            shape[3], shape[2],shape[1]);
    CUDA_POST_KERNEL_CHECK;

}

template <typename Dtype>
void RGB2XYZLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
  const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

    const Dtype* top_diff    = top[0]->cpu_diff();
    const Dtype* bottom_data = bottom[0]->cpu_data();
    Dtype* bottom_diff       = bottom[0]->mutable_cpu_diff();

    vector<int> shape = bottom[0]->shape();
    int npix = shape[0]*shape[2]*shape[3];
    RGB2XYZBackward<Dtype><<<CAFFE_GET_BLOCKS(npix), CAFFE_CUDA_NUM_THREADS>>>(
            npix, top_diff, bottom_diff, bottom_data, 
            shape[3], shape[2],shape[1]);
    CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(RGB2XYZLayer);

}  // namespace caffe
