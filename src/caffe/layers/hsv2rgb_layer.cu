#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>
#include <cmath>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/mgh_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void HSV2RGBForward(const int n, const Dtype* in, Dtype* out,
    int height, int width, int chans)
{
    CUDA_KERNEL_LOOP(index, n) {
        int blob_idx = index / (height*width);
        int px = index % (height*width);
        int src_idx = blob_idx * height*width*chans // blob n index
            + px; // pixel index
        int chan_stride = height*width;

        Dtype h = in[src_idx + 0*chan_stride];
        Dtype s = in[src_idx + 1*chan_stride];
        Dtype v = in[src_idx + 2*chan_stride];

        int hi = floor(h*6);
        Dtype f = h*6-hi;
        Dtype p = v*(1-s);
        Dtype q = v*(1-f*s);
        Dtype t = v*(1-(1-f)*s);

        Dtype r = 0;
        Dtype g = 0;
        Dtype b = 0;
        switch(hi%6) {
            case 0:
                r = v;
                g = t;
                b = p;
                break;
            case 1:
                r = q;
                g = v;
                b = p;
                break;
            case 2:
                r = p;
                g = v;
                b = t;
                break;
            case 3:
                r = p;
                g = q;
                b = v;
                break;
            case 4:
                r = t;
                g = p;
                b = v;
                break;
            case 5:
                r = v;
                g = p;
                b = q;
                break;
        }

        out[src_idx+0*chan_stride] = r;
        out[src_idx+1*chan_stride] = g;
        out[src_idx+2*chan_stride] = b;
    }
}

template <typename Dtype>
void HSV2RGBLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();

    vector<int> shape = bottom[0]->shape();
    int npix = shape[0]*shape[2]*shape[3];
    HSV2RGBForward<Dtype><<<CAFFE_GET_BLOCKS(npix), CAFFE_CUDA_NUM_THREADS>>>(
            npix, bottom_data, top_data, 
            shape[3], shape[2],shape[1]);
    CUDA_POST_KERNEL_CHECK;

}

INSTANTIATE_LAYER_GPU_FUNCS(HSV2RGBLayer);

}  // namespace caffe
